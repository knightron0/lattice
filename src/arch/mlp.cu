#include "hip/hip_runtime.h"
#include "mlp.cuh"
#include <stdio.h>

#define IN_FEATURES 784
#define OUT_FEATURES 10

Lattice (*activationFunctions[])(const Lattice&) = {relu, sigmoid, tanh, softmax};

MLP::MLP(int n_hidden, int* hidden_nodes, ActivationFunction* activations) {
  this->n_layers = n_hidden + 1;
  this->layers = (Linear**) malloc(n_hidden * sizeof(Linear*));
  for (int i = 0; i < this->n_layers; i++) {
    this->layers[i] = new Linear((i == 0) ? IN_FEATURES : hidden_nodes[i-1], (i == this->n_layers - 1) ? OUT_FEATURES : hidden_nodes[i], 1, (char *)"cuda");
  }

  this->activations = activations;
  
}

Lattice MLP::forward(Lattice x) {
  Lattice result = x;
  for (int i = 0; i < this->n_layers; i++) {
    result = this->layers[i]->forward(result);
    // result = activationFunctions[this->activations[i]](result);
    result.show(1, 0);
  }
  result.show(1, 1);
  printf("sumL %f\n", result.sum());
  result = softmax(result);
  result.show(1, 1);
  return result;
}