#include <lattice.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

void cpu_to_cuda(Lattice* lattice) {
  float *data_temp;

  hipMalloc((void **)&data_temp, lattice->kitna * sizeof(int));
  hipMemcpy(data_temp, lattice->data, lattice->data * sizeof(int), hipMemcpyHostToDevice);

  lattice->data = data_temp;

  lattice->kahan = (char*)malloc(strlen("cuda") + 1);
  strcpy(lattice->kahan, "cuda"); 
}

void cpu_to_cuda(Lattice* lattice) {
  float* data_tmp = (float*)malloc(lattice->kitna * sizeof(int));

  hipMemcpy(data_tmp, lattice->data, lattice->kitna * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(lattice->data);

  lattice->data = data_tmp;

  lattice->kahan = (char*)malloc(strlen("cpu") + 1);
  strcpy(lattice->kahan, "cpu"); 

  printf("Sent your lattice to: %s\n", lattice->kahan);
}