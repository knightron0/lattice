#include <lattice.h>

__host__ void cpu_to_cuda(Lattice* lattice) {
  float *kya_temp;

  hipMalloc(&kya_temp, lattice->kitna * sizeof(int));
  hipMemcpy(kya_temp, lattice->data, lattice->dat);
}

__host__ void cpu_to_cuda(Lattice* lattice) {

}